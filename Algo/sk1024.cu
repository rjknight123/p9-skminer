/*
* test routine for new algorithm
*
*/
#include "../hash/uint1024.h"
#include "hash/skein.h"
#include "hash/KeccakHash.h"
#include "miner.h"
#include "miner2.h"


extern int device_map[8];
extern void skein1024_setBlock(void *pdata, unsigned nHeight);
extern uint64_t skein1024_cpu_hash(int thr_id, int threads, uint64_t startNounce, int order, int threadsperblock = 256);
extern uint64_t sk1024_keccak_cpu_hash(int thr_id, int threads, uint64_t startNounce, uint64_t *d_nonceVector, uint64_t *d_hash, int order, int threadsperblock = 32);
extern void sk1024_keccak_cpu_init(int thr_id);
extern void sk1024_set_Target(const void *ptarget);

extern bool opt_benchmark;

extern bool scanhash_sk1024(unsigned int thr_id, uint32_t* TheData, uint1024 TheTarget, uint64_t &TheNonce, unsigned long long max_nonce, unsigned long long *hashes_done, int throughput, int thbpSkein, unsigned int nHeight)
{
	uint64_t *ptarget = (uint64_t*)&TheTarget;

	const uint64_t first_nonce = TheNonce;

	const uint64_t Htarg = ptarget[15];

	static bool init[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		hipDeviceReset();
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		sk1024_keccak_cpu_init(thr_id);

		init[thr_id] = true;
	}


	skein1024_setBlock((void*)TheData, nHeight);
	sk1024_set_Target(ptarget);

	int order = 0;
	uint64_t foundNonce = skein1024_cpu_hash(thr_id, throughput, ((uint64_t*)TheData)[26], order++, thbpSkein);
	if (foundNonce != 0xffffffffffffffff)
	{
		((uint64_t*)TheData)[26] = foundNonce;
		uint1024 skein;
		Skein1024_Ctxt_t ctx;
		Skein1024_Init(&ctx, 1024);
		Skein1024_Update(&ctx, (unsigned char *)TheData, 216);
		Skein1024_Final(&ctx, (unsigned char *)&skein);

		uint64_t keccak[16];
		Keccak_HashInstance ctx_keccak;
		Keccak_HashInitialize(&ctx_keccak, 576, 1024, 1024, 0x05);
		Keccak_HashUpdate(&ctx_keccak, (unsigned char *)&skein, 1024);
		Keccak_HashFinal(&ctx_keccak, (unsigned char *)&keccak);

		if (keccak[15] <= Htarg) {
			TheNonce = foundNonce; //return the nonce
			*hashes_done = foundNonce - first_nonce + 1;
			return true;
		}
		else {
			printf("GPU #%d: result for nonce %lu does not validate on CPU! \n", thr_id, foundNonce);
		}
	}
	((uint64_t*)TheData)[26] += throughput;

	uint64_t doneNonce = ((uint64_t*)TheData)[26];

	if (doneNonce < 18446744072149270489lu)
		*hashes_done = doneNonce - first_nonce + 1;

	return false;
}
