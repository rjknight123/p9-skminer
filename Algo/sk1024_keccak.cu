
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdint.h>
#include <memory.h>

extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);
__constant__ uint64_t pTarget[16];

#include "cuda_helper.h"
